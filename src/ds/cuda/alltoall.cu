#include "hip/hip_runtime.h"
#include "./alltoall.h"

#include <dmlc/logging.h>
#include <thread>
#define _CG_ABI_EXPERIMENTAL // enable experimental API
#include <hip/hip_cooperative_groups.h>

#include "../comm/comm_info.h"
#include "../utils.h"
#include "../../runtime/cuda/cuda_common.h"
#include "../context.h"
#include "./ds_kernel.h"
#include "../schedule.h"

using namespace dgl::runtime;
using namespace cooperative_groups;


namespace dgl {
namespace ds {

__device__
void sleep(int clock_count) {
  clock_t start_clock = clock();
  clock_t clock_offset = 0;
  while (clock_offset < clock_count) {
    clock_offset = clock() - start_clock;
  }

}

template<typename T> inline __device__
T vFetch(const volatile T* ptr) {
  return *ptr;
}

template<typename T> inline __device__
void vStore(volatile T* ptr, const T val) {
  *ptr = val;
}

class WaitFlag {
  volatile uint64_t *const flag;

 public:
  __host__ __device__ __forceinline__ WaitFlag(volatile uint64_t *const flag)
      : flag(flag) {
      }
  __device__ uint64_t get_flag() { return *flag; }
  __device__ __forceinline__ void init() { post(FLAG_INIT); }
  __device__ __forceinline__ void wait_init() { wait(FLAG_INIT); }
  __device__ __forceinline__ void unset() { post(FLAG_UNUSED); }
  __device__ __forceinline__ void wait_unset() { wait(FLAG_UNUSED); }
  
  __device__ __forceinline__ void wait(uint64_t val) {
    /*SPIN*/
    while ((*flag) != val) {
    }
  }
  __device__ __forceinline__ void post(uint64_t val) { *flag = val; }
  static constexpr uint64_t FLAG_INIT = ~0ull >> 1;
  static constexpr uint64_t FLAG_UNUSED = (~0ull >> 1) - 1;
};

struct CopyArgs {
  __host__ __device__ __forceinline__ CopyArgs(int tid, int n_threads,
                                               uint64_t *ready, uint64_t *done,
                                               uint64_t *next_ready,
                                               uint64_t *prev_done)
      : tid(tid),
        n_threads(n_threads),
        ready(ready),
        done(done),
        next_ready(next_ready),
        prev_done(prev_done) {}
  int tid, n_threads, group_size;
  int n_bytes;
  void *input, *output;
  void *my_recvbuff, *next_recvbuff;
  int send_size, recv_size;
  WaitFlag ready, done, next_ready, prev_done;
  
  int rank, peer_id;
};

#define DIVUP(x, y) ((x)+(y)-1)/(y)

template<typename T, int GroupSize>
__device__
void _Copy(CopyArgs args) {
  __shared__ experimental::block_tile_memory<8> shared;
  thread_block thb = experimental::this_thread_block(shared);
  auto thread_group = experimental::tiled_partition<GroupSize>(thb);

  constexpr int FETCH_BYTES = sizeof(T);
  constexpr int n_per_substage = RECV_BUFFER_SIZE / FETCH_BYTES;
  int send_size = args.send_size / FETCH_BYTES;
  int recv_size = args.recv_size / FETCH_BYTES;
  int send_substages = DIVUP(send_size, n_per_substage);
  int recv_substages = DIVUP(recv_size, n_per_substage);
  int n_substages = send_substages > recv_substages? send_substages:recv_substages;
  int bid = blockIdx.x;
  int local_tid = args.tid % args.group_size;
  if (local_tid == 0) {
    args.done.init();
    args.prev_done.wait_init();
    args.ready.init();
    args.next_ready.wait_init();
  }
  thread_group.sync();
  int send_ptr = args.tid;
  T* input = (T*)args.input;
  T* next_recvbuff = (T*)args.next_recvbuff;

  int recv_ptr = args.tid;
  T *my_recvbuff = (T*) args.my_recvbuff;
  T *output = (T*)args.output;

  for(int substage = 0; substage < n_substages; ++substage){
    int send_buff_ptr = local_tid;
    while(send_ptr < send_size && send_buff_ptr < n_per_substage) {
      T val = vFetch(input + send_ptr);
      vStore(next_recvbuff + send_buff_ptr, val);
      send_ptr += args.n_threads;
      send_buff_ptr += args.group_size;
    }
    thread_group.sync();
    __threadfence_system();
    if (local_tid == 0) {
      args.done.post(substage);
      args.prev_done.wait(substage);
    }
    thread_group.sync();
    int recv_buff_ptr = local_tid;
    while(recv_ptr < recv_size && recv_buff_ptr < n_per_substage) {
      T val = vFetch(my_recvbuff + recv_buff_ptr);
      vStore(output + recv_ptr, val);
      recv_ptr += args.n_threads;
      recv_buff_ptr += args.group_size;
    }
    thread_group.sync();
    if (local_tid == 0) {
      args.ready.post(substage);
      args.next_ready.wait(substage);
    }
    thread_group.sync();
  }
  if (local_tid == 0) {
    args.done.unset();
    args.prev_done.wait_unset();
    args.ready.unset();
    args.next_ready.wait_unset();
  }
  thread_group.sync();
}

// Deprecated
template<int GroupSize>
__device__
void _CopySendSize(int64_t* send_sizes, int64_t* recv_sizes, int peer_id, int local_tid, int n_threads_per_conn, ConnInfo* conn_info) {
  CopyArgs copy_args(local_tid, n_threads_per_conn, conn_info->my_ready, conn_info->my_done, conn_info->next_ready, conn_info->prev_done);
  copy_args.send_size = sizeof(int64_t);
  copy_args.recv_size = sizeof(int64_t);
  copy_args.group_size = n_threads_per_conn;
  copy_args.input = send_sizes + peer_id;
  copy_args.output = recv_sizes + peer_id;
  copy_args.my_recvbuff = conn_info->my_recv_buff;
  copy_args.next_recvbuff = conn_info->next_recv_buff;
  _Copy<int64_t, GroupSize>(copy_args);
}

template<typename T, int GroupSize>
__device__
void _CopyData(void* input, int64_t send_size, void* output, int64_t recv_size, int tid, int n_threads, int group_size, ConnInfo* conn_info, int rank, int peer_id) {
  CopyArgs copy_args(tid, n_threads, conn_info->my_ready, conn_info->my_done, conn_info->next_ready, conn_info->prev_done);
  copy_args.group_size = group_size;
  copy_args.send_size = send_size;
  copy_args.recv_size = recv_size;
  copy_args.input = input;
  copy_args.output = output;
  copy_args.my_recvbuff = conn_info->my_recv_buff;
  copy_args.next_recvbuff = conn_info->next_recv_buff;
  copy_args.rank = rank;
  copy_args.peer_id = peer_id;
  _Copy<T, GroupSize>(copy_args);
}

__device__ 
uint get_smid() {
  uint ret;
  asm("mov.u32 %0, %smid;" : "=r"(ret) );
  return ret;
}

template<typename T, bool exclusive, int GroupSize>
__global__
void _AlltoallKernel(AlltoallArgs args) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  if (tid == 0) {
    atomicAdd(args.cuda_launch_lock, -1);
  }
  int world_size = args.world_size;
  int rank = args.rank;
  int peer_id = tid / args.n_threads_per_conn;
  if(exclusive && peer_id >= rank) {
    peer_id++;
  }
  int local_tid = tid % args.n_threads_per_conn;
  ConnInfo* conn_info = args.comm_info->block_comm_info[bid].conn_info + peer_id;

  __shared__ IdType send_offset[9], recv_offset[9];
  if(tid <= args.world_size) {
    send_offset[tid] = args.send_offset == nullptr? tid: args.send_offset[tid];
    recv_offset[tid] = args.recv_offset == nullptr? tid: args.recv_offset[tid];
  }
  __syncthreads();

  void* sendbuff = (T*)args.sendbuff + send_offset[peer_id] * args.n_bytes / sizeof(T);
  void* recvbuff = (T*)args.recvbuff + recv_offset[peer_id] * args.n_bytes / sizeof(T);
  int64_t send_size = (send_offset[peer_id+1] - send_offset[peer_id]) * args.n_bytes;
  int64_t recv_size = (recv_offset[peer_id+1] - recv_offset[peer_id]) * args.n_bytes;
  int global_tid = bid * args.n_threads_per_conn + local_tid;
  _CopyData<T, GroupSize>(sendbuff, send_size, recvbuff, recv_size, global_tid, gridDim.x * args.n_threads_per_conn, args.n_threads_per_conn, conn_info, rank, peer_id);
}

__global__ 
void _DiffKernel(IdType* out, IdType* in, int size) {
  int tid = threadIdx.x;
  if(tid < size) {
    out[tid] = in[tid + 1] - in[tid];
  }
}

IdArray Diff(IdArray prefix_sum) {
  auto stream = CUDAThreadEntry::ThreadLocal()->stream;
  int size = prefix_sum->shape[0] - 1;
  IdArray ret = IdArray::Empty({size}, prefix_sum->dtype, prefix_sum->ctx);
  _DiffKernel<<<1, 32, 0, stream>>>(ret.Ptr<IdType>(), prefix_sum.Ptr<IdType>(), size);
  CUDACHECK(hipGetLastError());
  return ret;
}

#define ALLTOALL_SWITCH_ALIGN_SIZE(val, AlignType, ...) do {                 \
  if ((val) == 4) {                                             \
    using AlignType = int;                                      \
    {__VA_ARGS__}                                               \
  }                                                             \
  else if((val) == 8) {                                         \
    using AlignType = int64_t;                                  \
    {__VA_ARGS__}                                               \
  } else {                                                      \
    LOG(FATAL) << "Align size error";                           \
  }                                                             \
} while (0)

#define ALLTOALL_SWITCH_GROUP_SIZE(val, GroupSize, ...) do {                 \
  if ((val) == 16) {                                            \
    constexpr int GroupSize = 16;                               \
    {__VA_ARGS__}                                               \
  }                                                             \
  else if((val) == 64) {                                       \
    constexpr int GroupSize = 64;                              \
    {__VA_ARGS__}                                               \
  }                                                             \
  else if((val) == 128) {                                       \
    constexpr int GroupSize = 128;                              \
    {__VA_ARGS__}                                               \
  }                                                             \
  else if((val) == 256) {                                       \
    constexpr int GroupSize = 256;                              \
    {__VA_ARGS__}                                               \
  }                                                             \
  else if((val) == 512) {                                       \
    constexpr int GroupSize = 512;                              \
    {__VA_ARGS__}                                               \
  } else {                                                      \
    LOG(FATAL) << "Unsupport alltoall group size" << (val);     \
  }                                                             \
} while (0)

void CustomAlltoall(void* sendbuff, int64_t* send_offset, void* recvbuff, int64_t* recv_offset, int n_bytes, int align_size, CommInfo* comm_info, int rank, int world_size, int *cuda_launch_lock) {
  auto* thr_entry = CUDAThreadEntry::ThreadLocal();
  AlltoallArgs args;
  args.rank = rank;
  args.world_size = world_size;
  static constexpr int MAX_THREADS = 512;
  CHECK(MAX_THREADS % world_size == 0);
  args.n_threads_per_conn = MAX_THREADS / world_size;
  int n_threads = args.n_threads_per_conn * (world_size - 1);
  args.n_bytes = n_bytes;
  args.comm_info = comm_info->dev_comm_info;
  args.sendbuff = sendbuff;
  args.send_offset = send_offset;
  args.recvbuff = recvbuff;
  args.recv_offset = recv_offset;
  *cuda_launch_lock = comm_info->n_block;
  args.cuda_launch_lock = cuda_launch_lock;
  dim3 grid_dim(comm_info->n_block);
  dim3 block_dim(n_threads);
  void *kargs[] = {&args};
  ALLTOALL_SWITCH_ALIGN_SIZE(align_size, AlignType, {
    ALLTOALL_SWITCH_GROUP_SIZE(args.n_threads_per_conn, GroupSize, {
      CUDACHECK(hipLaunchKernel((void *)_AlltoallKernel<AlignType, true, GroupSize>,
                                      grid_dim, block_dim, kargs, 0, thr_entry->stream));
    });
  });

}

IdArray ExchangeSendSizes(IdArray send_offset, CommInfo* comm_info, int rank, int world_size, int *cuda_launch_lock) {
  auto stream = CUDAThreadEntry::ThreadLocal()->stream;
  auto send_sizes = Diff(send_offset);
  IdArray recv_sizes = IdArray::Empty({world_size}, send_offset->dtype, send_offset->ctx);

  AlltoallArgs args;
  args.rank = rank;
  args.world_size = world_size;
  args.n_threads_per_conn = 16;
  int n_threads = args.n_threads_per_conn * world_size;
  args.n_bytes = sizeof(IdType);
  args.comm_info = comm_info->dev_comm_info;
  args.sendbuff = send_sizes.Ptr<IdType>();
  args.send_offset = nullptr;
  args.recvbuff = recv_sizes.Ptr<IdType>();
  args.recv_offset = nullptr;
  dim3 grid_dim(1);
  dim3 block_dim(n_threads);
  *cuda_launch_lock = 1;
  args.cuda_launch_lock = cuda_launch_lock;
  void *kargs[] = {&args};
  ALLTOALL_SWITCH_ALIGN_SIZE(sizeof(IdType), AlignType, {
    ALLTOALL_SWITCH_GROUP_SIZE(args.n_threads_per_conn, GroupSize, {
      CUDACHECK(hipLaunchKernel((void *)_AlltoallKernel<AlignType, false, GroupSize>,
                                      grid_dim, block_dim, kargs, 0, stream));
    });
  });
  auto recv_offset = CumSum(recv_sizes, true);
  return recv_offset;
}

template <typename T, ncclDataType_t NCCL_DATA_TYPE>
void NCCLAllToAll(IdArray send_buffer, IdArray send_offset, IdArray recv_buffer, IdArray recv_offset, int expand_size, int rank, int world_size, ncclComm_t nccl_comm) {
  auto stream = CUDAThreadEntry::ThreadLocal()->stream;
  auto data_copy_stream = CUDAThreadEntry::ThreadLocal()->data_copy_stream;
  T* send_buffer_ptr = send_buffer.Ptr<T>();
  T* recv_buffer_ptr = recv_buffer.Ptr<T>();
  int type_bytes = sizeof(T);
  IdType* send_offset_ptr = send_offset.Ptr<IdType>();
  IdType* recv_offset_ptr = recv_offset.Ptr<IdType>();
  CUDACHECK(hipStreamSynchronize(stream));
  CUDACHECK(hipMemcpyAsync(recv_buffer_ptr + recv_offset_ptr[rank] * expand_size, 
                       send_buffer_ptr + send_offset_ptr[rank] * expand_size, 
                       (send_offset_ptr[rank + 1] - send_offset_ptr[rank]) * expand_size * type_bytes, hipMemcpyDeviceToDevice, data_copy_stream));
  ncclGroupStart();
  for(int r = 0; r < world_size; ++r) {
    if(r != rank) {
      IdType send_size = (send_offset_ptr[r+1] - send_offset_ptr[r]) * expand_size;
      IdType send_ptr = send_offset_ptr[r] * expand_size;
      IdType recv_size = (recv_offset_ptr[r+1] - recv_offset_ptr[r]) * expand_size;
      IdType recv_ptr = recv_offset_ptr[r] * expand_size;
      ncclSend(send_buffer_ptr + send_ptr, send_size, NCCL_DATA_TYPE, r, nccl_comm, stream);
      ncclRecv(recv_buffer_ptr + recv_ptr, recv_size, NCCL_DATA_TYPE, r, nccl_comm, stream);
    }
  }
  ncclGroupEnd();
}

std::pair<IdArray, IdArray> Alltoall(IdArray input, IdArray send_offset, int expand_size, int rank, int world_size) {
  auto* scheduler = Scheduler::Global();
  if(!GetEnvParam("USE_NCCL", 1)) {
    auto stream = CUDAThreadEntry::ThreadLocal()->stream;
    auto data_copy_stream = CUDAThreadEntry::ThreadLocal()->data_copy_stream;
    auto *ds_context = DSContext::Global();
    auto dgl_context = input->ctx;
    int type_bytes = input->dtype.bits / 8;
    int *cuda_launch_lock = &(CUDAThreadEntry::ThreadLocal()->cuda_launch_lock);
    int thread_id = CUDAThreadEntry::ThreadLocal()->thread_id;

    // NOTE: to guarantee the send_offset is ready
    CUDACHECK(hipStreamSynchronize(stream));
    auto host_send_offset = send_offset.CopyTo({kDLCPU, 0}, data_copy_stream);

    CommInfo *comm_info = ds_context->comm_info[thread_id].get();
    scheduler->TryComm(thread_id);
    auto recv_offset = ExchangeSendSizes(send_offset, comm_info, rank, world_size, cuda_launch_lock);
    //while (*cuda_launch_lock > 0);
    CUDACHECK(hipStreamSynchronize(stream));
    CHECK_EQ(*cuda_launch_lock, 0);
    scheduler->FinishComm();

    CUDACHECK(hipStreamSynchronize(stream));
    auto host_recv_offset = recv_offset.CopyTo({kDLCPU, 0}, stream);
    IdType total_recv_size = host_recv_offset.Ptr<IdType>()[world_size] * expand_size;
    auto recvbuff = IdArray::Empty({total_recv_size}, input->dtype, dgl_context);

    // Exclusive all to all
    if(world_size > 1) {
      scheduler->TryComm(thread_id);
      CustomAlltoall(input.Ptr<void>(), send_offset.Ptr<IdType>(), recvbuff.Ptr<void>(), recv_offset.Ptr<IdType>(), type_bytes * expand_size, input->dtype.bits / 8, comm_info, rank, world_size, cuda_launch_lock);
      //while (*cuda_launch_lock > 0);
      CUDACHECK(hipStreamSynchronize(stream));
      CHECK_EQ(*cuda_launch_lock, 0);
      scheduler->FinishComm();
    }

    // send data to myself in parallel
    auto* host_send_offset_ptr = host_send_offset.Ptr<IdType>();
    auto* host_recv_offset_ptr = host_recv_offset.Ptr<IdType>();

    int n_send_to_myself = host_send_offset_ptr[rank+1] - host_send_offset_ptr[rank];
    CUDACHECK(hipMemcpyAsync(recvbuff.Ptr<void>() + host_recv_offset_ptr[rank] * expand_size * type_bytes, input.Ptr<void>() + host_send_offset_ptr[rank] * expand_size * type_bytes, n_send_to_myself * type_bytes * expand_size, hipMemcpyDeviceToDevice, data_copy_stream));

    CUDACHECK(hipStreamSynchronize(stream));
    CUDACHECK(hipStreamSynchronize(data_copy_stream));
    return {recvbuff, recv_offset};
  } else {
    // NCCL
    CHECK(send_offset->dtype.bits == 64);
    auto stream = CUDAThreadEntry::ThreadLocal()->stream;
    auto data_copy_stream = CUDAThreadEntry::ThreadLocal()->data_copy_stream;
    auto dgl_context = input->ctx;
    auto *ds_context = DSContext::Global();
    auto host_dgl_context = DLContext{kDLCPU, 0};
    auto send_sizes = Diff(send_offset);
    int comm_token = CUDAThreadEntry::ThreadLocal()->thread_id;
    IdArray recv_sizes = IdArray::Empty({world_size}, send_offset->dtype, dgl_context);
    IdArray range_seq = Range(0, world_size + 1, 64, host_dgl_context);
    int thread_id = CUDAThreadEntry::ThreadLocal()->thread_id;
    ncclComm_t nccl_comm = ds_context->nccl_comm[thread_id];

    scheduler->TryComm(thread_id);
    NCCLAllToAll<int64_t, ncclInt64>(send_sizes, range_seq, recv_sizes, range_seq, 1, rank, world_size, nccl_comm);
    CUDACHECK(hipStreamSynchronize(stream));
    scheduler->FinishComm();

    auto host_send_offset = send_offset.CopyTo(host_dgl_context, stream);
    CUDACHECK(hipStreamSynchronize(data_copy_stream));
    auto recv_offset = CumSum(recv_sizes, true);
    CUDACHECK(hipStreamSynchronize(stream));
    IdArray host_recv_offset = recv_offset.CopyTo(host_dgl_context, stream);
    auto* host_recv_offset_ptr = host_recv_offset.Ptr<IdType>();
    int n_recv = host_recv_offset_ptr[world_size] * expand_size;
    auto recvbuff = IdArray::Empty({n_recv}, input->dtype, dgl_context);

    scheduler->TryComm(thread_id);
    if(input->dtype.bits == 32) {
      NCCLAllToAll<int, ncclInt32>(input, host_send_offset, recvbuff, host_recv_offset, expand_size, rank, world_size, nccl_comm);
    } else {
      NCCLAllToAll<int64_t, ncclInt64>(input, host_send_offset, recvbuff, host_recv_offset, expand_size, rank, world_size, nccl_comm);
    }
    CUDACHECK(hipStreamSynchronize(stream));
    scheduler->FinishComm();
    CUDACHECK(hipStreamSynchronize(data_copy_stream));
    return {recvbuff, recv_offset};
  }
}

}
}
