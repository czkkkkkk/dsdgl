#include "hip/hip_runtime.h"
#include "./alltoall.h"

#include <dmlc/logging.h>
#include <thread>

#include "../comm/comm_info.h"
#include "../utils.h"
#include "../../runtime/cuda/cuda_common.h"
#include "../context.h"
#include "./ds_kernel.h"
#include "../schedule.h"

using namespace dgl::runtime;

namespace dgl {
namespace ds {

// 500 MB
static constexpr int MAX_RECV_BUFFER_SIZE = 500 * 1024 * 1024;

__device__
void sleep(int clock_count) {
  clock_t start_clock = clock();
  clock_t clock_offset = 0;
  while (clock_offset < clock_count) {
    clock_offset = clock() - start_clock;
  }

}

template<typename T> inline __device__
T vFetch(const volatile T* ptr) {
  return *ptr;
}

template<typename T> inline __device__
void vStore(volatile T* ptr, const T val) {
  *ptr = val;
}

class WaitFlag {
  volatile uint64_t *const flag;

 public:
  __host__ __device__ __forceinline__ WaitFlag(volatile uint64_t *const flag)
      : flag(flag) {
      }
  __device__ uint64_t get_flag() { return *flag; }
  __device__ __forceinline__ void unset() { post(FLAG_UNUSED); }
  __device__ __forceinline__ void wait_unset() { wait(FLAG_UNUSED); }
  
  __device__ __forceinline__ void wait(uint64_t val) {
    /*SPIN*/
    while ((*flag) != val) {
    }
  }
  __device__ __forceinline__ void post(uint64_t val) { *flag = val; }
  const static uint64_t FLAG_UNUSED = ~0ull >> 1;
};

struct CopyArgs {
  __host__ __device__ __forceinline__ CopyArgs(int tid, int n_threads,
                                               uint64_t *ready, uint64_t *done,
                                               uint64_t *next_ready,
                                               uint64_t *prev_done)
      : tid(tid),
        n_threads(n_threads),
        ready(ready),
        done(done),
        next_ready(next_ready),
        prev_done(prev_done) {}
  int tid, n_threads, group_size;
  int n_bytes;
  void *input, *output;
  void *my_recvbuff, *next_recvbuff;
  int send_size, recv_size;
  WaitFlag ready, done, next_ready, prev_done;
};

template<typename T>
__device__
void _Copy(CopyArgs args) {
  static const int FETCH_BYTES = sizeof(T);
  int bid = blockIdx.x;
  if (args.tid % args.group_size == 0) {
    args.ready.post(1);
    args.next_ready.wait(1);
  }
  __syncthreads();
  int tid = args.tid;
  int buff_ptr = args.tid % args.group_size;
  int send_size = args.send_size / FETCH_BYTES;
  T* input = (T*)args.input;
  T* next_recvbuff = (T*)args.next_recvbuff;
  while(tid < send_size) {
    T val = vFetch(input + tid);
    vStore(next_recvbuff + buff_ptr, val);
    // args.next_recvbuff[buff_ptr] = args.input[tid];
    tid += args.n_threads;
    buff_ptr += args.group_size;
  }
  __threadfence_system();
  __syncthreads();
  if (args.tid % args.group_size == 0) {
    args.done.post(1);
    args.prev_done.wait(1);
  }
  __syncthreads();

  // ------- Receive -----------
  tid = args.tid;
  buff_ptr = args.tid % args.group_size;
  int recv_size = args.recv_size / FETCH_BYTES;
  T *my_recvbuff = (T*) args.my_recvbuff;
  T *output = (T*)args.output;
  while(tid < recv_size) {
    T val = vFetch(my_recvbuff + buff_ptr);
    vStore(output + tid, val);
    // args.output[tid] = args.my_recvbuff[buff_ptr]; 
    tid += args.n_threads;
    buff_ptr += args.group_size;
  }
  __syncthreads();
  if (args.tid % args.group_size == 0) {
    args.ready.unset();
    args.next_ready.wait_unset();
    args.done.unset();
    args.prev_done.wait_unset();
  }
  __syncthreads();
}

__device__
void _CopySendSize(int64_t* send_sizes, int64_t* recv_sizes, int peer_id, int local_tid, int n_threads_per_conn, ConnInfo* conn_info) {
  CopyArgs copy_args(local_tid, n_threads_per_conn, conn_info->my_ready, conn_info->my_done, conn_info->next_ready, conn_info->prev_done);
  copy_args.send_size = sizeof(int64_t);
  copy_args.recv_size = sizeof(int64_t);
  copy_args.group_size = n_threads_per_conn;
  copy_args.input = send_sizes + peer_id;
  copy_args.output = recv_sizes + peer_id;
  copy_args.my_recvbuff = conn_info->my_recv_buff;
  copy_args.next_recvbuff = conn_info->next_recv_buff;
  _Copy<int64_t>(copy_args);
}

template<typename T>
__device__
void _CopyData(void* input, int64_t send_size, void* output, int64_t recv_size, int tid, int n_threads, int group_size, ConnInfo* conn_info) {
  CopyArgs copy_args(tid, n_threads, conn_info->my_ready, conn_info->my_done, conn_info->next_ready, conn_info->prev_done);
  copy_args.group_size = group_size;
  copy_args.send_size = send_size;
  copy_args.recv_size = recv_size;
  copy_args.input = input;
  copy_args.output = output;
  copy_args.my_recvbuff = conn_info->my_recv_buff;
  copy_args.next_recvbuff = conn_info->next_recv_buff;
  _Copy<T>(copy_args);
}

__device__ 
uint get_smid() {
  uint ret;
  asm("mov.u32 %0, %smid;" : "=r"(ret) );
  return ret;
}

template<typename T, bool exclusive>
__global__
void _AlltoallKernel(AlltoallArgs args) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  int world_size = args.world_size;
  int rank = args.rank;
  int peer_id = tid / args.n_threads_per_conn;
  if(exclusive && peer_id >= rank) {
    peer_id++;
  }
  int local_tid = tid % args.n_threads_per_conn;
  ConnInfo* conn_info = args.comm_info->block_comm_info[bid].conn_info + peer_id;

  __shared__ IdType send_offset[9], recv_offset[9];
  if(tid <= args.world_size) {
    send_offset[tid] = args.send_offset == nullptr? tid: args.send_offset[tid];
    recv_offset[tid] = args.recv_offset == nullptr? tid: args.recv_offset[tid];
  }
  __syncthreads();

  void* sendbuff = (T*)args.sendbuff + send_offset[peer_id] * args.n_bytes / sizeof(T);
  void* recvbuff = (T*)args.recvbuff + recv_offset[peer_id] * args.n_bytes / sizeof(T);
  int64_t send_size = (send_offset[peer_id+1] - send_offset[peer_id]) * args.n_bytes;
  int64_t recv_size = (recv_offset[peer_id+1] - recv_offset[peer_id]) * args.n_bytes;
  int global_tid = bid * args.n_threads_per_conn + local_tid;
  _CopyData<T>(sendbuff, send_size, recvbuff, recv_size, global_tid, gridDim.x * args.n_threads_per_conn, args.n_threads_per_conn, conn_info);
}

__global__ 
void _DiffKernel(IdType* out, IdType* in, int size) {
  int tid = threadIdx.x;
  if(tid < size) {
    out[tid] = in[tid + 1] - in[tid];
  }
}

IdArray Diff(IdArray prefix_sum) {
  auto stream = CUDAThreadEntry::ThreadLocal()->stream;
  int size = prefix_sum->shape[0] - 1;
  IdArray ret = IdArray::Empty({size}, prefix_sum->dtype, prefix_sum->ctx);
  _DiffKernel<<<1, 32, 0, stream>>>(ret.Ptr<IdType>(), prefix_sum.Ptr<IdType>(), size);
  CUDACHECK(hipGetLastError());
  return ret;
}

void CustomAlltoall(void* sendbuff, int64_t* send_offset, void* recvbuff, int64_t* recv_offset, int n_bytes, int align_size, CommInfo* comm_info, int rank, int world_size) {
  auto* thr_entry = CUDAThreadEntry::ThreadLocal();
  AlltoallArgs args;
  args.rank = rank;
  args.world_size = world_size;
  static constexpr int MAX_THREADS = 1024;
  CHECK(MAX_THREADS % world_size == 0);
  args.n_threads_per_conn = MAX_THREADS / world_size;
  int n_threads = args.n_threads_per_conn * (world_size - 1);
  args.n_bytes = n_bytes;
  args.comm_info = comm_info->dev_comm_info;
  args.sendbuff = sendbuff;
  args.send_offset = send_offset;
  args.recvbuff = recvbuff;
  args.recv_offset = recv_offset;
  dim3 grid_dim(comm_info->n_block);
  dim3 block_dim(n_threads);
  void *kargs[] = {&args};
  hipError_t e;
  if(align_size == 4) {
    e = hipLaunchKernel((void *)_AlltoallKernel<int, true>,
                                    grid_dim, block_dim, kargs, 0, thr_entry->stream);
  } else if(align_size == 8) {
    CHECK(n_bytes % 8 == 0);
    e = hipLaunchKernel((void *)_AlltoallKernel<int64_t, true>,
                                    grid_dim, block_dim, kargs, 0, thr_entry->stream);
  } else {
    LOG(FATAL) << "Unsupported bytes: " << n_bytes;
  }

  CUDACHECKERR(e);
}

IdArray ExchangeSendSizes(IdArray send_offset, CommInfo* comm_info, int rank, int world_size) {
  auto stream = CUDAThreadEntry::ThreadLocal()->stream;
  auto send_sizes = Diff(send_offset);
  IdArray recv_sizes = IdArray::Empty({world_size}, send_offset->dtype, send_offset->ctx);

  AlltoallArgs args;
  args.rank = rank;
  args.world_size = world_size;
  static constexpr int MAX_THREADS = 1024;
  CHECK(MAX_THREADS % world_size == 0);
  args.n_threads_per_conn = MAX_THREADS / world_size;
  int n_threads = args.n_threads_per_conn * world_size;
  // args.n_threads_per_conn = 16;
  // int n_threads = args.n_threads_per_conn * world_size;
  args.n_bytes = sizeof(IdType);
  args.comm_info = comm_info->dev_comm_info;
  args.sendbuff = send_sizes.Ptr<IdType>();
  args.send_offset = nullptr;
  args.recvbuff = recv_sizes.Ptr<IdType>();
  args.recv_offset = nullptr;
  dim3 grid_dim(1);
  dim3 block_dim(n_threads);
  void *kargs[] = {&args};
  CUDACHECK(hipLaunchKernel((void *)_AlltoallKernel<IdType, false>,
                                  grid_dim, block_dim, kargs, 0, stream));
  hipStreamSynchronize(stream);
  auto recv_offset = CumSum(recv_sizes, true);
  return recv_offset;
}

template <typename T, ncclDataType_t NCCL_DATA_TYPE>
void NCCLAllToAll(IdArray send_buffer, IdArray send_offset, IdArray recv_buffer, IdArray recv_offset, int expand_size, int rank, int world_size, ncclComm_t nccl_comm) {
  auto stream = CUDAThreadEntry::ThreadLocal()->stream;
  auto data_copy_stream = CUDAThreadEntry::ThreadLocal()->data_copy_stream;
  T* send_buffer_ptr = send_buffer.Ptr<T>();
  T* recv_buffer_ptr = recv_buffer.Ptr<T>();
  int type_bytes = sizeof(T);
  IdType* send_offset_ptr = send_offset.Ptr<IdType>();
  IdType* recv_offset_ptr = recv_offset.Ptr<IdType>();
  CUDACHECK(hipStreamSynchronize(stream));
  CUDACHECK(hipMemcpyAsync(recv_buffer_ptr + recv_offset_ptr[rank] * expand_size, 
                       send_buffer_ptr + send_offset_ptr[rank] * expand_size, 
                       (send_offset_ptr[rank + 1] - send_offset_ptr[rank]) * expand_size * type_bytes, hipMemcpyDeviceToDevice, data_copy_stream));
  ncclGroupStart();
  for(int r = 0; r < world_size; ++r) {
    if(r != rank) {
      IdType send_size = (send_offset_ptr[r+1] - send_offset_ptr[r]) * expand_size;
      IdType send_ptr = send_offset_ptr[r] * expand_size;
      IdType recv_size = (recv_offset_ptr[r+1] - recv_offset_ptr[r]) * expand_size;
      IdType recv_ptr = recv_offset_ptr[r] * expand_size;
      ncclSend(send_buffer_ptr + send_ptr, send_size, NCCL_DATA_TYPE, r, nccl_comm, stream);
      ncclRecv(recv_buffer_ptr + recv_ptr, recv_size, NCCL_DATA_TYPE, r, nccl_comm, stream);
    }
  }
  ncclGroupEnd();
}

std::pair<IdArray, IdArray> Alltoall(IdArray input, IdArray send_offset, int expand_size, int rank, int world_size, ncclComm_t nccl_comm, bool is_sample) {
  auto* scheduler = Scheduler::Global();
  int comm_token;
  if (is_sample) {
    comm_token = CUDAThreadEntry::ThreadLocal()->thread_id;
  } else {
    comm_token = CUDAThreadEntry::ThreadLocal()->thread_id + DSContext::Global()->sample_worker_num;
  }

  if(!GetEnvParam("USE_NCCL", 1)) {
    auto stream = CUDAThreadEntry::ThreadLocal()->stream;
    auto data_copy_stream = CUDAThreadEntry::ThreadLocal()->data_copy_stream;
    auto* ds_context = DSContext::Global();
    auto dgl_context = input->ctx;
    int type_bytes = input->dtype.bits / 8;
    auto recvbuff = IdArray::Empty({MAX_RECV_BUFFER_SIZE / type_bytes}, input->dtype, dgl_context);

    // NOTE: to guarantee the send_offset is ready
    CUDACHECK(hipStreamSynchronize(stream));
    auto host_send_offset = send_offset.CopyTo({kDLCPU, 0}, data_copy_stream);

    CommInfo *comm_info;
    if (is_sample) {
      comm_info = &ds_context->comm_info;
    } else {
      comm_info = &ds_context->comm_info_load;
    }

    scheduler->TryComm(comm_token);
    auto recv_offset = ExchangeSendSizes(send_offset, comm_info, rank, world_size);
    scheduler->FinishComm();
    CUDACHECK(hipStreamSynchronize(stream));

    // Exclusive all to all
    if(world_size > 1) {
      scheduler->TryComm(comm_token);
      CustomAlltoall(input.Ptr<void>(), send_offset.Ptr<IdType>(), recvbuff.Ptr<void>(), recv_offset.Ptr<IdType>(), type_bytes * expand_size, input->dtype.bits / 8, comm_info, rank, world_size);
      if (is_sample) {
        printf("rank: %d finish send sample data\n", rank);
      } else {
        printf("rank: %d finish send load data\n", rank);
      }
      scheduler->FinishComm();
    }

    // send data to myself in parallel
    auto host_recv_offset = recv_offset.CopyTo({kDLCPU, 0}, data_copy_stream);
    CUDACHECK(hipStreamSynchronize(data_copy_stream));
    auto* host_send_offset_ptr = host_send_offset.Ptr<IdType>();
    auto* host_recv_offset_ptr = host_recv_offset.Ptr<IdType>();
    CHECK_LE(host_recv_offset_ptr[world_size] * expand_size * input->dtype.bits / 8, MAX_RECV_BUFFER_SIZE);

    int n_send_to_myself = host_send_offset_ptr[rank+1] - host_send_offset_ptr[rank];
    CUDACHECK(hipMemcpyAsync(recvbuff.Ptr<void>() + host_recv_offset_ptr[rank] * expand_size * type_bytes, input.Ptr<void>() + host_send_offset_ptr[rank] * expand_size * type_bytes, n_send_to_myself * type_bytes * expand_size, hipMemcpyDeviceToDevice, data_copy_stream));

    CUDACHECK(hipStreamSynchronize(stream));
    CUDACHECK(hipStreamSynchronize(data_copy_stream));
    recvbuff = recvbuff.CreateView({(signed long) host_recv_offset_ptr[world_size] * expand_size}, input->dtype);
    return {recvbuff, recv_offset};
  } else {
    // NCCL
    CHECK(nccl_comm != nullptr);
    CHECK(send_offset->dtype.bits == 64);
    auto stream = CUDAThreadEntry::ThreadLocal()->stream;
    auto data_copy_stream = CUDAThreadEntry::ThreadLocal()->data_copy_stream;
    auto dgl_context = input->ctx;
    auto host_dgl_context = DLContext{kDLCPU, 0};
    auto send_sizes = Diff(send_offset);
    IdArray recv_sizes = IdArray::Empty({world_size}, send_offset->dtype, dgl_context);
    IdArray range_seq = Range(0, world_size + 1, 64, host_dgl_context);
    
    scheduler->TryComm(comm_token);
    NCCLAllToAll<int64_t, ncclInt64>(send_sizes, range_seq, recv_sizes, range_seq, 1, rank, world_size, nccl_comm);
    //CUDACHECK(hipStreamSynchronize(stream));
    scheduler->FinishComm();

    auto host_send_offset = send_offset.CopyTo(host_dgl_context, stream);
    CUDACHECK(hipStreamSynchronize(data_copy_stream));
    auto recv_offset = CumSum(recv_sizes, true);
    CUDACHECK(hipStreamSynchronize(stream));
    IdArray host_recv_offset = recv_offset.CopyTo(host_dgl_context, stream);
    auto* host_recv_offset_ptr = host_recv_offset.Ptr<IdType>();
    int n_recv = host_recv_offset_ptr[world_size] * expand_size;
    auto recvbuff = IdArray::Empty({n_recv}, input->dtype, dgl_context);

    scheduler->TryComm(comm_token);
    if(input->dtype.bits == 32) {
      NCCLAllToAll<int, ncclInt32>(input, host_send_offset, recvbuff, host_recv_offset, expand_size, rank, world_size, nccl_comm);
    } else {
      NCCLAllToAll<int64_t, ncclInt64>(input, host_send_offset, recvbuff, host_recv_offset, expand_size, rank, world_size, nccl_comm);
    }
    //CUDACHECK(hipStreamSynchronize(stream));
    scheduler->FinishComm();
    CUDACHECK(hipStreamSynchronize(data_copy_stream));
    return {recvbuff, recv_offset};
  }
}

}
}
